#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <openssl/aes.h>

#define BLOCK_SIZE 16

__global__ void AES_encrypt_cuda( unsigned char *d_plaintext, unsigned char *d_ciphertext, AES_KEY *d_enc_key, int num_blocks ){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if( idx < num_blocks ){
        AES_encrypt( &d_plaintext[idx * BLOCK_SIZE], &d_ciphertext[idx * BLOCK_SIZE], d_enc_key );
    }
}

void AES_encrypt_parallel( const std::vector<unsigned char> &plaintext, std::vector<unsigned char> &ciphertext, const AES_KEY &enc_key ){
    int num_blocks = plaintext.size() / BLOCK_SIZE;
    ciphertext.resize( plaintext.size() );
    
    unsigned char *d_plaintext, *d_ciphertext;
    AES_KEY *d_enc_key;
    
    hipMalloc( &d_plaintext, plaintext.size() );
    hipMalloc( &d_ciphertext, ciphertext.size() );
    hipMalloc( &d_enc_key, sizeof( AES_KEY ) );
    
    hipMemcpy( d_plaintext, plaintext.data(), plaintext.size(), hipMemcpyHostToDevice );
    hipMemcpy( d_enc_key, &enc_key, sizeof( AES_KEY ), hipMemcpyHostToDevice );
    
    int threadsPerBlock = 256;
    int blocksPerGrid = ( num_blocks + threadsPerBlock - 1 ) / threadsPerBlock;
    AES_encrypt_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_plaintext, d_ciphertext, d_enc_key, num_blocks);
    
    hipMemcpy( ciphertext.data(), d_ciphertext, ciphertext.size(), hipMemcpyDeviceToHost );
    
    hipFree( d_plaintext );
    hipFree( d_ciphertext );
    hipFree( d_enc_key );
}

int main(){
    const unsigned char key[BLOCK_SIZE] = "0123456789abcdef";
    const unsigned char plaintext[] = "This is a secret message that needs encryption.";
    
    int padded_size = ( ( sizeof( plaintext ) + BLOCK_SIZE - 1 ) / BLOCK_SIZE ) * BLOCK_SIZE;
    std::vector<unsigned char> padded_plaintext( padded_size, 0 );
    memcpy( padded_plaintext.data(), plaintext, sizeof( plaintext ) );
    
    AES_KEY enc_key;
    AES_set_encrypt_key( key, 128, &enc_key );
    
    std::vector<unsigned char> ciphertext;
    AES_encrypt_parallel( padded_plaintext, ciphertext, enc_key );
    
    std::cout << "Encryption complete." << std::endl;
    return 0;
}
